#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>
#include <hip/hip_runtime_api.h> //In samples/common/inc

#include "loadSpins.h"
#include "layers.h"
#include "sampleCorrelate.h"
#include "workingMemory.h"
#include "types.h"

#define DBUG //Save stuff to files

#define SIGN(x) ((x > 0) ? 1.f: -1.f)
#ifndef MIN
#define MIN(a, b) ((a > b) ? b : a)
#endif
#ifndef MAX
#define MAX(a, b) ((a > b) ? a : b)
#endif
#define IDX2F(i,j,ld) (((j)*(ld))+(i))

#define THREADS_PER 128

__global__
void weightMatrixUpdate(float *d_W, float *d_previousWstep, 
		        float *d_modelCorrelations, float *d_dataCorrelations, 
			float lr, float mom, float sparsity,
			int batchSize, int N_h, int N_v);

int main(int argc, char **argv){

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0){
        fprintf(stderr, "Error: no CUDA supporting devices.\n");
	exit(EXIT_FAILURE);
    }
    int dev = 0; 
    hipSetDevice(dev);
    
    const char *printMSG = "Incorrect number of arguments: Usage: \n\
			    ./curbm filename N_visible N_hidden k_samples batchsize epochs lr mom sparsity\n";
    if (argc < 10){
        printf("%s", printMSG);
	return 0;
    }
    else if (argc > 10){
        printf("%s", printMSG);
        return 0;
    }

    char *filename = argv[1];
    int N_v = atoi(argv[2]);
    int N_h = atoi(argv[3]);
    int k = atoi(argv[4]);
    int batchSize = atoi(argv[5]);
    int epochs = atoi(argv[6]);
    float lr = atof(argv[7]);
    float mom = atof(argv[8]);
    float sparsity = atof(argv[9]);
  
    int Nbits = 0, numSamples = 0;
    float *h_spinList = loadSpins(filename, &Nbits);
    if (h_spinList == NULL){
        printf("Exiting.\n");
	return 0;	   
    }
    numSamples = Nbits / N_v;

    Layer visible, hidden;
    DataCorrContainer container;
    float *h_W, *d_W, *d_previousWstep;
    float *h_modelCorrelations, *d_modelCorrelations, *d_random;
    float *h_dataCorrelations, *d_dataCorrelations;
    allocateLayer(&visible, N_v, k);
    allocateLayer(&hidden, N_h, k);
    allocateCorrContainer(&container, N_v, N_h, batchSize);
    allocateMemory(&h_W, &d_W, &d_previousWstep,
		   &h_modelCorrelations, &d_modelCorrelations,
		   &h_dataCorrelations, &d_dataCorrelations,
		   &d_random, N_v, N_h);
   
    hipStream_t stream1, stream2;
    checkCudaErrors(hipStreamCreate(&stream1));
    checkCudaErrors(hipStreamCreate(&stream2)); 
    //cuBLAS init
    hipblasHandle_t cublasHandle1, cublasHandle2;
    checkCudaErrors(hipblasCreate(&cublasHandle1));
    checkCudaErrors(hipblasCreate(&cublasHandle2));
    checkCudaErrors(hipblasSetStream(cublasHandle1, stream1));
    checkCudaErrors(hipblasSetStream(cublasHandle2, stream2));
    //cuRAND init
    hiprandGenerator_t rng1, rng2;
    checkCudaErrors(hiprandCreateGenerator(&rng1, HIPRAND_RNG_PSEUDO_DEFAULT));
    checkCudaErrors(hiprandSetStream(rng1, stream1));
    checkCudaErrors(hiprandSetPseudoRandomGeneratorSeed(rng1, 920989ULL));
    checkCudaErrors(hiprandCreateGenerator(&rng2, HIPRAND_RNG_PSEUDO_DEFAULT));
    checkCudaErrors(hiprandSetStream(rng2, stream2));
    checkCudaErrors(hiprandSetPseudoRandomGeneratorSeed(rng2, 14859ULL));
    
    //initialize Weights
    float stdDev = 1.f/sqrt( (float) N_h); 
    checkCudaErrors(hiprandGenerateNormal(rng1, d_W, (size_t) N_v*N_h, 0.f, stdDev));

    FILE *fp_saveW = fopen("W.dat", "w");
    checkCudaErrors(hipMemcpy(h_W, d_W, sizeof(float)*N_v*N_h, hipMemcpyDeviceToHost));
    // Save weights 
    for (int i=0; i < N_v; i++){
        fprintf(fp_saveW, "\n");
        for (int j=0; j < N_h; j++){
            fprintf(fp_saveW, "%f\t", h_W[IDX2F(i,j, N_v)]);
        }
    } //Saving initial W

    //Time measurement
    hipEvent_t start, stop;
    float time;
    checkCudaErrors(hipEventCreate(&start)); checkCudaErrors(hipEventCreate(&stop));

    float *d_initialVisible, *h_spinPtr = h_spinList;
    checkCudaErrors(hipMalloc(&d_initialVisible, visible.BYTES));

    //Start timer
    checkCudaErrors(hipEventRecord(start, 0));

    dim3 blocks(ceil((float) (N_v * N_h)/(float) THREADS_PER), 1, 1);
    dim3 threads(THREADS_PER, 1, 1);
    int numBatches = ceil((float) numSamples / (float) batchSize);

    //epochs = 1; numBatches = 1;
    FILE *fpConv = fopen("meanEnergyPerBatch.dat","w");
    
    //epochs = 1; numBatches = 2;

    printf("Performing %d epochs with  %d batches\n", epochs, numBatches);
    for (int ep = 0; ep < epochs; ep++){
        for (int i = 0; i < numBatches; i++){ 
	    
	    int startGibbs = MIN(numSamples-1, ceil((rand()/(float)RAND_MAX) * numSamples));
/*Stream1*/ checkCudaErrors(hipMemcpyAsync(d_initialVisible, h_spinList + N_v*startGibbs, 
        			            visible.BYTES, hipMemcpyHostToDevice, stream1));
/*Stream2*/ checkCudaErrors(hipMemcpyAsync(container.d_visibleBatch, h_spinPtr, visible.BYTES * batchSize, 
            			            hipMemcpyHostToDevice, stream2));
            h_spinPtr = h_spinList + MIN(N_v * batchSize, numSamples - batchSize - 1);
            
/*Stream1*/ computeK_Gibbs(visible, hidden, d_W, d_initialVisible, d_random, cublasHandle1, rng1);
            computeModelCorrelations(visible, hidden, d_modelCorrelations, cublasHandle1);
            
/*Stream2*/ float energy = computeDataCorrelations(d_dataCorrelations, d_W, container, cublasHandle2, rng2);
            fprintf(fpConv, "%f\n", energy); 
            //Wait for both to finish before updating weight matrix 
            checkCudaErrors(hipStreamQuery(stream1)); checkCudaErrors(hipStreamQuery(stream2));
	    weightMatrixUpdate<<<blocks, threads, 0, stream1>>>(d_W, d_previousWstep,
			                                        d_modelCorrelations, d_dataCorrelations, 
            		                                        lr, mom, sparsity, batchSize, N_h, N_v);
        }
    }
   
    //Stop timer
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    fclose(fpConv);

    hipEventElapsedTime(&time, start, stop);    
    printf("Elapsed time: %f ms\n", time);

    checkCudaErrors(hipMemcpy(h_W, d_W, sizeof(float)*N_v*N_h, hipMemcpyDeviceToHost));
    // Save weights 
    for (int i=0; i < N_v; i++){
        fprintf(fp_saveW, "\n");
        for (int j=0; j < N_h; j++){
            fprintf(fp_saveW, "%f\t", h_W[IDX2F(i,j, N_v)]);
        }
    }
    fclose(fp_saveW);


#ifdef DBUG
 
    copyLayerDeviceToHost(&visible);
    copyLayerDeviceToHost(&hidden);
    checkCudaErrors(hipMemcpy(h_modelCorrelations, d_modelCorrelations, 
			       sizeof(float)*N_v*N_h, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_dataCorrelations, d_dataCorrelations, 
			       sizeof(float)*N_v*N_h, hipMemcpyDeviceToHost));
    
    FILE *fpW = fopen("dbugW.dat", "w");
    FILE *fph = fopen("dbugHidden.dat", "w");
    FILE *fpv = fopen("dbugVisible.dat", "w");
    
    for (int i=0; i < N_v; i++){
	if (i % N_v == 0)
	    fprintf(fpv, "\n");
        fprintf(fpv, "%f\t", h_spinList[i]);
    }
    //printf("model correlations = ");
    for (int i=0; i < N_v; i++){
        fprintf(fpW, "\n");
        for (int j=0; j < N_h; j++){
            fprintf(fpW, "%f\t", h_W[IDX2F(i,j, N_v)]);
        }
    }
    for (int i=0; i < N_v; i++){
        fprintf(fpW, "\n");
        for (int j=0; j < N_h; j++){
            fprintf(fpW, "%f\t", h_modelCorrelations[IDX2F(i,j, N_v)]);
        }
    }
    for (int i=0; i < N_v; i++){
        fprintf(fpW, "\n");
        for (int j=0; j < N_h; j++){
            fprintf(fpW, "%f\t", h_dataCorrelations[IDX2F(i,j, N_v)]);
        }
    }
    //printf("\nHidden sample = ");
    int nhiddens = hidden.kSamples * N_h;
    for (int j=0; j < nhiddens; j++){
	if (j % N_h ==0)
	    fprintf(fph, "\n");
	fprintf(fph, "%f\t", hidden.h_samples[j]);
    }
    //printf("\nHidden Conditional Probability = ");
    for (int j=0; j < N_h; j++){
	if (j % N_h ==0)
	    fprintf(fph, "\n");
	fprintf(fph, "%f\t", hidden.h_conditionalP[j]);
    }
    //printf("\nHidden Energies = ");
    for (int j=0; j < N_h; j++){
	if (j % N_h ==0)
	    fprintf(fph, "\n");
	fprintf(fph, "%f\t", hidden.h_energySum[j]);
    }
    int nvisibles = visible.kSamples * N_v;
    //printf("\nVisible sample = ");
    for (int i=0; i < nvisibles; i++){
	if (i % N_v == 0){
	    fprintf(fpv, "\n");
	}
	fprintf(fpv, "%f\t", visible.h_samples[i]);
    }
    //printf("\nVisible Conditional Probability = ");
    for (int i=0; i < N_v; i++){
	if (i % N_v == 0){
	    fprintf(fpv, "\n");
	}
	fprintf(fpv, "%f\t", visible.h_conditionalP[i]);
    }
    //printf("\nVisible energies = ");
    for (int i=0; i < N_v; i++){
	if (i % N_v == 0){
	    fprintf(fpv, "\n");
	}
	fprintf(fpv, "%f\t", visible.h_energySum[i]);
    }
    fclose(fpW);
    fclose(fph);
    fclose(fpv);

#endif
   
    // Clean up 
    checkCudaErrors(hipblasDestroy(cublasHandle1));
    checkCudaErrors(hipblasDestroy(cublasHandle2));
    checkCudaErrors(hiprandDestroyGenerator(rng1));
    checkCudaErrors(hiprandDestroyGenerator(rng2));
    checkCudaErrors(hipStreamDestroy(stream1));
    checkCudaErrors(hipStreamDestroy(stream2));

    freeLayer(visible); freeLayer(hidden);
    freeCorrContainer(container); 
    freeMemory(&h_W, &d_W, &d_previousWstep, 
	       &h_modelCorrelations, &d_modelCorrelations,
	       &h_dataCorrelations, &d_dataCorrelations,
	       &d_random);

    return EXIT_SUCCESS;
}


__global__
void weightMatrixUpdate(float *d_W, float *d_previousWstep,
		        float *d_modelCorrelations,
		        float *d_dataCorrelations, 
			float lr, float mom, float sparsity,
			int batchSize, int N_h, int N_v){
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid >= N_h * N_v){
        return;
    }
    float W = d_W[tid];
    float lastW = d_previousWstep[tid];
    float corrDiff = (d_dataCorrelations[tid] - d_modelCorrelations[tid]);
    float CDstep = (lr / ((float) batchSize)) * corrDiff;
    float L1step = -1.f * sparsity * SIGN(W);
    float newStep = (1.f-mom)*CDstep + mom*lastW + L1step;
    d_previousWstep[tid] = newStep; //update previous steps
    d_W[tid] = W + newStep; 
}


