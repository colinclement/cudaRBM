#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>
#include <hip/hip_runtime_api.h> //In samples/common/inc

#include "loadSpins.h"
#include "layers.h"
#include "sampleCorrelate.h"
#include "workingMemory.h"
#include "types.h"

//#define DBUG //Save stuff to files

#ifndef MIN
#define MIN(a, b) ((a > b) ? b : a)
#endif
#ifndef MAX
#define MAX(a, b) ((a > b) ? a : b)
#endif
#define IDX2F(i,j,ld) (((j)*(ld))+(i))

#define THREADS_PER 64

__global__
void weightMatrixUpdate(float *d_W, float *d_modelCorrelations,
		        float *d_dataCorrelations, 
			float lr, float mom, float sparsity,
			int N_h, int N_v);

int main(int argc, char **argv){

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0){
        fprintf(stderr, "Error: no CUDA supporting devices.\n");
	exit(EXIT_FAILURE);
    }
    int dev = 0; 
    hipSetDevice(dev);
    
    const char *printMSG = "Incorrect number of arguments: Usage: \n\
			    ./curbm filename N_visible N_hidden k_samples batchsize epochs lr mom sparsity\n";
    if (argc < 10){
        printf("%s", printMSG);
	return 0;
    }
    else if (argc > 10){
        printf("%s", printMSG);
        return 0;
    }

    char *filename = argv[1];
    int N_v = atoi(argv[2]);
    int N_h = atoi(argv[3]);
    int k = atoi(argv[4]);
    int batchSize = atoi(argv[5]);
    int epochs = atoi(argv[6]);
    float lr = atof(argv[7]);
    float mom = atof(argv[8]);
    float sparsity = atof(argv[9]);
  
    int Nbits = 0, numSamples = 0;
    float *h_spinList = loadSpins(filename, &Nbits);
    if (h_spinList == NULL){
        printf("Exiting.\n");
	return 0;	   
    }
    numSamples = Nbits / N_v;

    Layer visible, hidden;
    DataCorrContainer container;
    float *h_W, *d_W;
    float *h_modelCorrelations, *d_modelCorrelations, *d_random;
    float *h_dataCorrelations, *d_dataCorrelations;
    allocateLayer(&visible, N_v, k);
    allocateLayer(&hidden, N_h, k);
    allocateCorrContainer(&container, N_v, N_h, batchSize);
    allocateMemory(&h_W, &d_W,
		   &h_modelCorrelations, &d_modelCorrelations,
		   &h_dataCorrelations, &d_dataCorrelations,
		   &d_random, N_v, N_h);
    
    //cuBLAS init
    hipblasHandle_t cublasHandle;
    checkCudaErrors(hipblasCreate(&cublasHandle));
    //cuRAND init
    hiprandGenerator_t rng;
    checkCudaErrors(hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT));
    checkCudaErrors(hiprandSetPseudoRandomGeneratorSeed(rng, 920989ULL));
    //initialize Weights 
    checkCudaErrors(hiprandGenerateNormal(rng, d_W, (size_t) N_v*N_h, 0.f, 0.05f));
   
    //Time measurement
    hipEvent_t start, stop;
    float time;
    checkCudaErrors(hipEventCreate(&start)); checkCudaErrors(hipEventCreate(&stop));

    float *d_initialVisible, *h_spinPtr = h_spinList;
    checkCudaErrors(hipMalloc(&d_initialVisible, visible.BYTES));

    //Start timer
    checkCudaErrors(hipEventRecord(start, 0));

    dim3 blocks(ceil((float) (N_v * N_h)/(float) THREADS_PER), 1, 1);
    dim3 threads(THREADS_PER, 1, 1);
    int numBatches = ceil((float) numSamples / (float) batchSize);

    printf("Only doing 5 batches for profiling\n");
    for (int ep = 0; ep < epochs; ep++){
        for (int i = 0; i < 1; i++){ 
            int startGibbs = ceil((rand()/(float)RAND_MAX) * numSamples);
            checkCudaErrors(hipMemcpy(d_initialVisible, h_spinList + N_v*startGibbs, 
        			           visible.BYTES, hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(container.d_visibleBatch, h_spinPtr, visible.BYTES * batchSize, 
            			   hipMemcpyHostToDevice));
            h_spinPtr += MIN(N_v * batchSize, numSamples - batchSize - 1);
            
            computeK_Gibbs(visible, hidden, d_W, d_initialVisible, d_random, cublasHandle, rng);
            computeModelCorrelations(visible, hidden, d_modelCorrelations, cublasHandle);
            computeDataCorrelations(d_dataCorrelations, d_W, container, cublasHandle, rng);

            weightMatrixUpdate<<<blocks, threads>>>(d_W, d_modelCorrelations,
            	                                d_dataCorrelations, 
            		                        lr, mom, sparsity, N_h, N_v);
            checkCudaErrors(hipDeviceSynchronize());
        }
        h_spinPtr = h_spinList;
    }

    //Stop timer
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));

    hipEventElapsedTime(&time, start, stop);    
    printf("Elapsed time: %f ms\n", time);

    checkCudaErrors(hipMemcpy(h_W, d_W, sizeof(float)*N_v*N_h, hipMemcpyDeviceToHost));
    // Save weights 
    FILE *fp_saveW = fopen("W.dat", "w");
    for (int i=0; i < N_v; i++){
        fprintf(fp_saveW, "\n");
        for (int j=0; j < N_h; j++){
            fprintf(fp_saveW, "%f\t", h_W[IDX2F(i,j, N_v)]);
        }
    }
    fclose(fp_saveW);


#ifdef DBUG
 
    copyLayerDeviceToHost(&visible);
    copyLayerDeviceToHost(&hidden);
    checkCudaErrors(hipMemcpy(h_modelCorrelations, d_modelCorrelations, 
			       sizeof(float)*N_v*N_h, hipMemcpyDeviceToHost));
    
    FILE *fpW = fopen("dbugW.dat", "w");
    FILE *fph = fopen("dbugHidden.dat", "w");
    FILE *fpv = fopen("dbugVisible.dat", "w");
    
    printf("first spin configuration:\n");
    for (int i=0; i < N_v; i++){
	if (i % N_v == 0)
	    fprintf(fpv, "\n");
        fprintf(fpv, "%f\t", h_spinList[i]);
    }
    //printf("model correlations = ");
    for (int i=0; i < N_v; i++){
        fprintf(fpW, "\n");
        for (int j=0; j < N_h; j++){
            fprintf(fpW, "%f\t", h_W[IDX2F(i,j, N_v)]);
        }
    }
    for (int i=0; i < N_v; i++){
        fprintf(fpW, "\n");
        for (int j=0; j < N_h; j++){
            fprintf(fpW, "%f\t", h_modelCorrelations[IDX2F(i,j, N_v)]);
        }
    }
    //printf("\nHidden sample = ");
    int nhiddens = hidden.kSamples * N_h;
    for (int j=0; j < nhiddens; j++){
	if (j % N_h ==0)
	    fprintf(fph, "\n");
	fprintf(fph, "%f\t", hidden.h_samples[j]);
    }
    //printf("\nHidden Conditional Probability = ");
    for (int j=0; j < N_h; j++){
	if (j % N_h ==0)
	    fprintf(fph, "\n");
	fprintf(fph, "%f\t", hidden.h_conditionalP[j]);
    }
    //printf("\nHidden Energies = ");
    for (int j=0; j < N_h; j++){
	if (j % N_h ==0)
	    fprintf(fph, "\n");
	fprintf(fph, "%f\t", hidden.h_energySum[j]);
    }
    int nvisibles = visible.kSamples * N_v;
    //printf("\nVisible sample = ");
    for (int i=0; i < nvisibles; i++){
	if (i % N_v == 0){
	    fprintf(fpv, "\n");
	}
	fprintf(fpv, "%f\t", visible.h_samples[i]);
    }
    //printf("\nVisible Conditional Probability = ");
    for (int i=0; i < N_v; i++){
	if (i % N_v == 0){
	    fprintf(fpv, "\n");
	}
	fprintf(fpv, "%f\t", visible.h_conditionalP[i]);
    }
    //printf("\nVisible energies = ");
    for (int i=0; i < N_v; i++){
	if (i % N_v == 0){
	    fprintf(fpv, "\n");
	}
	fprintf(fpv, "%f\t", visible.h_energySum[i]);
    }
    fclose(fpW);
    fclose(fph);
    fclose(fpv);

#endif
   
    // Clean up 
    checkCudaErrors(hipblasDestroy(cublasHandle));
    checkCudaErrors(hiprandDestroyGenerator(rng));

    freeLayer(visible); freeLayer(hidden);
    freeCorrContainer(container); 
    freeMemory(&h_W, &d_W, 
	       &h_modelCorrelations, &d_modelCorrelations,
	       &h_dataCorrelations, &d_dataCorrelations,
	       &d_random);

    return EXIT_SUCCESS;
}

__global__
void weightMatrixUpdate(float *d_W, float *d_modelCorrelations,
		        float *d_dataCorrelations, 
			float lr, float mom, float sparsity,
			int N_h, int N_v){
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid < N_h * N_v){
        return;
    }
    float dw = d_W[tid];
    float wUpdate = (d_dataCorrelations[tid] - d_modelCorrelations[tid]);
    d_W[tid] = mom * dw + lr * (1 - mom) * wUpdate - sparsity * fabs(dw);
}


