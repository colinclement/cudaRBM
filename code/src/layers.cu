#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "types.h"

__host__
void copyLayerDeviceToHost(Layer unitLayer){
    checkCudaErrors(hipMemcpy(unitLayer.h_samples, unitLayer.d_samples, 
	            unitLayer.SAMPLEBYTES, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(unitLayer.h_conditionalP, unitLayer.d_conditionalP, 
	            unitLayer.BYTES, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(unitLayer.h_energySum, unitLayer.d_energySum, 
	            unitLayer.BYTES, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(unitLayer.h_random, unitLayer.d_random, 
	            unitLayer.BYTES, hipMemcpyDeviceToHost));
}

__host__
Layer allocateLayer(int N_units, int numSamples){
    Layer newLayer;
    int BYTES = N_units * sizeof(float);
    newLayer.BYTES = BYTES;
    newLayer.SAMPLEBYTES = BYTES * numSamples;
    newLayer.N_units = N_units;
    newLayer.numSamples = numSamples;
    newLayer.h_samples = (float *)malloc(BYTES * numSamples);
    memset(newLayer.h_samples, 0, BYTES * numSamples);
    newLayer.h_conditionalP = (float *)malloc(BYTES);
    memset(newLayer.h_conditionalP, 0, BYTES);
    newLayer.h_energySum = (float *)malloc(BYTES);
    memset(newLayer.h_energySum, 0, BYTES);
    newLayer.h_random = (float *)malloc(BYTES);
    memset(newLayer.h_random, 0, BYTES);
  
    checkCudaErrors(hipMalloc((void **)&newLayer.d_samples, newLayer.SAMPLEBYTES));
    checkCudaErrors(hipMemset(newLayer.d_samples, 0, newLayer.SAMPLEBYTES));

    newLayer.d_samplePtr = newLayer.d_samples; //Start ptr at beginning

    checkCudaErrors(hipMalloc((void **)&newLayer.d_random, BYTES));
    checkCudaErrors(hipMemset(newLayer.d_random, 0, BYTES));
    checkCudaErrors(hipMalloc((void **)&newLayer.d_conditionalP, BYTES));
    checkCudaErrors(hipMemset(newLayer.d_conditionalP, 0, BYTES));
    checkCudaErrors(hipMalloc((void **)&newLayer.d_energySum, BYTES));
    checkCudaErrors(hipMemset(newLayer.d_energySum, 0, BYTES));

    return newLayer;
}

__host__
void updateLayerSample(Layer unitLayer, float *h_hostSamples, int  BYTES,
                       hipStream_t stream){
    checkCudaErrors(hipMemcpyAsync(unitLayer.d_samples, h_hostSamples, BYTES,
                                    hipMemcpyHostToDevice, stream));
}

__host__
void freeLayer(Layer newLayer){
    free(newLayer.h_samples); newLayer.h_samples=NULL;
    hipFree(newLayer.d_samples); newLayer.d_samples=NULL;
    free(newLayer.h_conditionalP); newLayer.h_conditionalP=NULL;
    hipFree(newLayer.d_random); newLayer.d_random=NULL;
    hipFree(newLayer.d_conditionalP); newLayer.d_conditionalP=NULL;
    free(newLayer.h_energySum); newLayer.h_energySum=NULL;
    hipFree(newLayer.d_energySum); newLayer.d_energySum=NULL;
    free(newLayer.h_random); newLayer.h_random = NULL;
}

