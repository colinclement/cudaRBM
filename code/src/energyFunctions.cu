#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>

#include "types.h"

#ifndef MIN
#define MIN(a, b) ((a > b) ? b : a)
#endif
#ifndef MAX
#define MAX(a, b) ((a > b) ? a : b)
#endif
#define IDX2F(i,j,ld) (((j)*(ld))+(i))

__host__
void allToAll(Layer sampleLayer, Layer givenLayer,
              const float *d_W, hipStream_t stream,
              hipblasHandle_t handle){
    int sN = sampleLayer.N_units, gN = givenLayer.N_units;
    int N_v = MAX(sN, gN), N_h = MIN(sN, gN);
    float a = 1.f, beta = 0.f;
    hipblasOperation_t OP = ((sN > gN) ? HIPBLAS_OP_N : HIPBLAS_OP_T);
    checkCudaErrors(hipblasSgemv(handle, OP, N_v, N_h, &a, d_W, N_v, 
	          	   	givenLayer.d_samplePtr, 1, &beta, 
                    sampleLayer.d_energySum, 1));
}

__host__
void convolution(Layer sampleLayer, Layer givenLayer,
                 const float *d_W, hipStream_t stream,
                 hipblasHandle_t handle){
    return;
}

