#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>

#include "types.h"

#ifndef MIN
#define MIN(a, b) ((a > b) ? b : a)
#endif
#ifndef MAX
#define MAX(a, b) ((a > b) ? a : b)
#endif
#define IDX2F(i,j,ld) (((j)*(ld))+(i))

__host__
void allToAll(Layer sampleLayer, Layer givenLayer,
              Connection conn, hipStream_t stream,
              hipblasHandle_t handle){
    int N_v = conn.fan_in, N_h = conn.fan_out;
    float a = 1.f, beta = 0.f;
    hipblasOperation_t OP; 
    OP = ((givenLayer.N_units == conn.cols) ? HIPBLAS_OP_N : HIPBLAS_OP_T);
    checkCudaErrors(hipblasSgemv(handle, OP, N_v, N_h, &a, conn.d_W, N_v, 
	          	   	givenLayer.d_samplePtr, 1, &beta, 
                    sampleLayer.d_energySum, 1));
}

__host__
void convolution(Layer sampleLayer, Layer givenLayer,
                 Connection conn, hipStream_t stream,
                 hipblasHandle_t handle){
    return;
}

__global__
void partialEnergyConvolution(Layer givenLayer, Connection conn){
    return;
}

