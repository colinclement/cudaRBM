#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>
#include <hip/hip_runtime_api.h>

#include "types.h"

//#define DBUG_K
//#define DBUG_GIBBS

//Note this is really sig(-x)
#define sig(x) (1.f/(1.f + expf(x)))
#define THREADS_PER 32
#ifndef MIN
#define MIN(a, b) ((a > b) ? b : a)
#endif
#ifndef MAX
#define MAX(a, b) ((a > b) ? a : b)
#endif
#define IDX2F(i,j,ld) (((j)*(ld))+(i))

__global__
void sampleConditional(Layer unitLayer, const int N_units){
    /*   samples conditional probability of visible (hidden) units
     *          unitLayer : an instance of Layer (hidden or visible)
     *		d_random : uniform (0,1] random numbers
     * */
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid >= N_units){
        return;
    }
    float P_unit_is_1 = sig(-2.f * unitLayer.d_energySum[tid]);
    //unitLayer.d_conditionalP[tid] = P_unit_is_1;
    float rnd = unitLayer.d_random[tid]; 
    unitLayer.d_samplePtr[tid] = 2.f*((float)(P_unit_is_1 > rnd))-1.f;
}

__host__
void computeGibbsSample(Layer sampleLayer, Layer givenLayer,
                        const float *d_W, energyFunc energy,
                        hipStream_t stream, hipblasHandle_t handle){
                        //hipStream_t stream, hipblasHandle_t handle){
    // Sample state of sampleLayer given state of givenLayer
    // NOTE: Assumes visible Layer has MORE units than hidden Layer!!!
    int sN = sampleLayer.N_units;
    //float a = -2.f, beta = 0.f;//minus in E instead of in sigmoid
    //int N_v = MAX(sN, gN), N_h = MIN(sN, gN);
    //hipblasOperation_t OP = ((sN > gN) ? HIPBLAS_OP_N : HIPBLAS_OP_T);
    dim3 blocks((int) ceilf((float) sN / (float) THREADS_PER), 1, 1);
    dim3 threads(THREADS_PER, 1, 1);
    //Can put energy functional here
    //Energy functional should take two layers (sample and given),
    //And should populate sample.d_energySum
    //checkCudaErrors(hipblasSgemv(handle, OP, N_v, N_h, &a, d_W, N_v, 
	//          	   	givenLayer.d_samplePtr, 1, &beta, 
    //                sampleLayer.d_energySum, 1));
    energy(sampleLayer, givenLayer, d_W, stream, handle);
    sampleConditional<<<blocks, threads, 0, stream>>>(sampleLayer, sN);
}

__host__
void computeKGibbs(Layer visible, Layer hidden,
                   const float *d_W, energyFunc energy, 
                   float *d_random, hiprandGenerator_t rng,
                   hipStream_t stream, hipblasHandle_t handle){
    int N_v = visible.N_units, N_h = hidden.N_units;    
    visible.d_samplePtr = visible.d_samples; 
    for (int i=0; i < visible.numSamples; i++){
        checkCudaErrors(hiprandGenerateUniform(rng, d_random, N_v+N_h));
        visible.d_random = d_random; hidden.d_random = d_random + N_v;
        hidden.d_samplePtr = hidden.d_samples + i * N_h;
        computeGibbsSample(hidden, visible, d_W, energy, stream, handle);
        visible.d_samplePtr = visible.d_samples + i * N_v;
        computeGibbsSample(visible, hidden, d_W, energy, stream, handle);
    } 
    visible.d_samplePtr = visible.d_samples;//Reset moving pointer
    hidden.d_samplePtr = hidden.d_samples;
}

__host__
void computeGibbsGivenData(Layer visible, Layer hidden,
                           float *d_W, energyFunc energy, 
                           hiprandGenerator_t rng, 
                           hipStream_t stream, hipblasHandle_t handle){
    int N_v = visible.N_units, N_h = hidden.N_units;    
    for (int i = 0; i < visible.numSamples; i++){
        checkCudaErrors(hiprandGenerateUniform(rng, hidden.d_random, N_h));
        hidden.d_samplePtr = hidden.d_samples + i * N_h;
        visible.d_samplePtr = visible.d_samples + i * N_v;
        computeGibbsSample(hidden, visible, d_W, energy, stream, handle);
    }
}

__host__
void computeCorrelations(Layer visible, Layer hidden,
		                 float *d_correlations, hipblasHandle_t handle){
    int k = visible.numSamples, N_v = visible.N_units, N_h = hidden.N_units;
    const float alpha = 1.f/((float) k), beta = 0.f;
    checkCudaErrors(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
			        N_v, N_h, k, &alpha, visible.d_samples, N_v, 
				    hidden.d_samples, N_h, &beta, d_correlations, N_v));
}


//TODO: Investigate function pointers for improved modularity in transitioning to CRBM

//TODO: Consider updating weight matrix in place (probabably a lot more efficient).
//      Also consider concurrent updating and sampling.

