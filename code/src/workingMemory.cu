#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


void allocateMemory(float **d_previousWstep, float **d_random, 
                    int N_v, int N_h){
    int WBYTES = N_v * N_h * sizeof(float);
    checkCudaErrors(hipMalloc((void **)d_previousWstep, WBYTES));
    checkCudaErrors(hipMemset(*d_previousWstep, 0, WBYTES));
    checkCudaErrors(hipMalloc((void **)d_random, sizeof(float)*(N_v+N_h)));
    checkCudaErrors(hipMemset(*d_random, 0, sizeof(float)*(N_v+N_h)));
}

void freeMemory(float *d_previousWstep, float *d_random){
    hipFree(d_previousWstep); hipFree(d_random);
    d_previousWstep = NULL; d_random = NULL;
}
