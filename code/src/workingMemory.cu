#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


void allocateMemory(float **h_W, float **d_W, 
		    float **h_modelCorrelations, float **d_modelCorrelations,
		    float **h_dataCorrelations, float **d_dataCorrelations,
		    float **d_random, int N_v, int N_h){
    int WBYTES = N_v * N_h * sizeof(float);
    int HBYTES = N_h * sizeof(float);
    *h_W = (float *)malloc(WBYTES);
    memset(*h_W, 0, WBYTES);
    //These things can probably go after debugging is done
    *h_modelCorrelations = (float *)malloc(WBYTES);
    memset(*h_modelCorrelations, 0, WBYTES);
    *h_dataCorrelations = (float *)malloc(HBYTES);
    memset(*h_dataCorrelations, 0, HBYTES);

    checkCudaErrors(hipMalloc((void **)d_W, WBYTES));
    checkCudaErrors(hipMemset(*d_W, 0, WBYTES));
    checkCudaErrors(hipMalloc((void **)d_modelCorrelations, WBYTES));
    checkCudaErrors(hipMemset(*d_modelCorrelations, 0, WBYTES));
    checkCudaErrors(hipMalloc((void **)d_dataCorrelations, WBYTES));
    checkCudaErrors(hipMemset(*d_dataCorrelations, 0, WBYTES));
    
    checkCudaErrors(hipMalloc((void **)d_random, sizeof(float)*(N_v+N_h)));
    checkCudaErrors(hipMemset(*d_random, 0, sizeof(float)*(N_v+N_h)));
}

void freeMemory(float **h_W, float **d_W,
		float **h_modelCorrelations, float **d_modelCorrelations,
		float **h_dataCorrelations, float **d_dataCorrelations,
		float **d_random){
    free(*h_W); *h_W = NULL;
    free(*h_modelCorrelations), *h_modelCorrelations = NULL; 
    hipFree(d_W); hipFree(d_random); hipFree(d_modelCorrelations);
    hipFree(d_dataCorrelations); 
    *d_W = NULL; *d_random = NULL; 
    *d_modelCorrelations = NULL; *d_dataCorrelations = NULL; 
}
